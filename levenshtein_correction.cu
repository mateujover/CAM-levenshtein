
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>
#include <clocale>
#include <cctype>

#define HASH_SIZE 20
#define PADDING_CHAR '\0'

//Macro extaida de exemplos de NVIDIA: modificado printf -> cout
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    cout << "Error at " << __FILE__ << " line: " << __LINE__ << endl; \
    return EXIT_FAILURE;}} while(0)

using namespace std;

// Definimos una funcion minimo por si CUDA/C++ no lo incluye para GPU
__device__ unsigned short my_strlen(char *s)
{
	unsigned short len = 0;
	while(len < HASH_SIZE && s[len] != PADDING_CHAR) { len++; }
	return len;
}

__device__ unsigned short minimo(unsigned short a, unsigned short b){
	if(a > b)
		return b;
	else
		return a;
}

__device__ unsigned short lev_dist(char *s1, char *s2){
	unsigned short l1, l2, i, j, c, res, w;
    l1 = my_strlen(s1);
    l2 = my_strlen(s2);
	// Verifica que exista algo que comparar
    if (l1 == 0) return(l2);
    if (l2 == 0) return(l1);
    w = l1 + 1;
	// Reserva matriz con malloc: m[i,j] = m[j*w+i] !!
    unsigned short m[((HASH_SIZE+1)*HASH_SIZE+1) + HASH_SIZE+1];
	// Rellena primera fila y primera columna
    for (i = 0; i <= l1; i++) m[i] = i;
    for (j = 0; j <= l2; j++) m[j*w] = j;
	// Recorremos resto de la matriz llenando pesos
    for (i = 1; i <= l1; i++){
		for (j = 1; j <= l2; j++)
		{ 
			if (s1[i-1] == s2[j-1]) 
				c = 0;
			else 
				c = 1;
			
		    m[j*w+i] = minimo(minimo(m[j*w+i-1]+1, m[(j-1)*w+i]+1), m[(j-1)*w+i-1]+c);
		}
	}
	// Devolvemos esquina final de la matriz
    res = m[l2*w+l1];
    return(res);
}

/*----------------------------------------------------------------------------
  ----------------------------------------------------------------------------
  -----------------------  GLOBAL FUNCTIONS: KERNELS  ------------------------
  ----------------------------------------------------------------------------
  ----------------------------------------------------------------------------*/

__global__ void k_setupPadding(char *first_word, unsigned int total_entradas, char _padd = (char)PADDING_CHAR){
	
	unsigned int idx = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned int stride = blockDim.x * gridDim.x;
	
	while(idx < total_entradas){ // Aqui direccionamos de caracter en caracter
		for(unsigned char _off = 0; _off < HASH_SIZE; _off++){
			first_word[(idx * HASH_SIZE) + _off] = _padd;
		}
		idx += stride;
	}
	
}

__global__ void k_levenshtein(char *str, char *first_word, unsigned int total_entradas, unsigned int *out_idx, unsigned short *out_dist)
{
	char local_str[HASH_SIZE];
	// Bottleneck
	for(unsigned char i = 0; i < HASH_SIZE; i++) {
		local_str[i] = str[i];
	}
	// Para copiar de vuelta en CPU
	unsigned short local_min = 0xFFFF;
	unsigned int min_idx = 0;
	// Limites e indices
	unsigned int idx = threadIdx.x + (blockDim.x * blockIdx.x); // Id para stride 0
	unsigned int stride = blockDim.x * gridDim.x;
	// Valores calculados de cada thread
    unsigned short local_dist;
	
	while(idx < total_entradas){
		// Calculo de distancia:
		local_dist = lev_dist( local_str, first_word + (idx * HASH_SIZE * sizeof(char)) );
		// Actualizar valores:
		if(local_dist < local_min){
			local_min = local_dist;
			min_idx = idx;
		}
		// Siguiente palabra:
		idx += stride; // Numero de palabra
		
	} // End while busqueda del thread
	
	// Copia de resultados para cada thread de los minimos, back to CPU...
	out_dist[threadIdx.x + (blockDim.x * blockIdx.x)] = local_min;
	out_idx[threadIdx.x + (blockDim.x * blockIdx.x)] = min_idx;
}

/*---------------------------------------------------------------------------
 ----------------------------------------------------------------------------
 ------------------------------  MAIN PROGRAM  ------------------------------
 ----------------------------------------------------------------------------
 ---------------------------------------------------------------------------*/

void correct_usage();

int main(int argc, char **argv)
{	
    unsigned int threads_per_block = 256; // MAX THREADS PER BLOCKS
	unsigned int num_blocks = 4;
	enum input_t {performance_mode, correct_mode} mode;
	mode = performance_mode;
	// Command line inputs
	if( argc == 2 && strcmp(argv[1], "--help") == 0 )
	{
		correct_usage();
		return 0;
		cout << "Not ended!!!!!!!!" << endl;
	}
	if(argc >= 4 && strcmp(argv[1], "--grid") == 0)
	{
		threads_per_block = stoi(argv[2]);
		num_blocks = stoi(argv[3]);
	}
	if(argc == 5) 
	{
		if(strcmp(argv[4], "--correct") == 0){
			mode = correct_mode; 
		} // else ja considerat
	}
	unsigned short out_gpu_len = num_blocks * threads_per_block;
    
	if(strcmp(setlocale(LC_ALL, NULL), "C") == 0)
	{
		cout << "Tenim encoding 'C'. Canvi a nes des sistema..." << endl;
		if(setlocale(LC_ALL, "") == NULL)
		{
			cout << "Failed to set new locale\nEXIT";
			return EXIT_FAILURE;
		}
	}
	
	ifstream fitxer;
	cout << "Obrir diccionari..." << endl;
	fitxer.open("dictionary.txt");
		
	string str;
	unsigned int entradas = 0; // Maximo elementos: 4,294,967,296 -> [0, 4,294,967,296)
	
	if(!fitxer){
		cout << endl << "Diccionari no obert" << endl << "EXIT" << endl;
		return EXIT_FAILURE;	
	}
	else { // Lectura de la cantidad de entradas en el fichero:
		unsigned short longest = 0;
		cout << endl << "Diccionari obert" << endl << "Llegint..." << endl;
		getline(fitxer, str); // En caso de solo tener una entrada o poder detectar los ficheros de solo una entrada.
		while( !fitxer.eof() ){
			entradas++;
			getline(fitxer, str);
			if(longest < str.length()){
				longest = str.length();
			}
		}
		cout << "Entrades diccionari: " << entradas << endl;
		cout << "Palabra mas larga: " << longest << endl;
		if(longest > HASH_SIZE){
			cout << endl << "Macro HASH_SIZE insuficient. Minim recomanat: " << longest << endl;
			cout << "Exit" << endl;
			return EXIT_FAILURE;
		}
	}
	cout << endl << "Close file...  " << endl;
	fitxer.close();
		
	// Allotjam espai per guardar tot el diccionari. Per això tenim el nombre d'entrades
	// Llegim fixer i guardam les línies sense completar amb PADDING_CHAR -> Això ho passam a kernel...
	cout << "Allotjar memoria diccionari per " << entradas << " entrades de 25 char..." << endl;
	char *first_word;
	CUDA_CALL( hipMallocManaged(&first_word, entradas * HASH_SIZE * sizeof(char)) );
	
	// Cridada kernel de inicialitzacio memoria GPU -> PADDING_CHARs
	//
	cout << endl << "Kernel call. Threads per block: " << threads_per_block << endl;
	cout << "k_setupPadding..." << endl;
	k_setupPadding<<<num_blocks, threads_per_block>>>(first_word, entradas);
	CUDA_CALL( hipDeviceSynchronize() ); 
	cout << "Padding ended" << endl;
	
	// Cambio de puntero on el que trabajar para no perder origen
	char *word = first_word;

	// Lectura de fitxer i bolcat de chars
	//
	cout << endl << "Reopen file" << endl;
	fitxer.open("dictionary.txt");
	if(!fitxer){
		cout << endl << "Fitxer no obert" << endl << "EXIT" << endl;
		return EXIT_FAILURE;
	}
	cout << "Fitxer a inici" << endl;

	cout << endl << "Lectura de fitxer a memoria..." << endl;
	unsigned short i;
	while( !fitxer.eof() ){// Llegir línies senceres, despres convertir a nes nostro format de string
		getline(fitxer, str);
		i = 0;
		
		while(i < str.length()){ // Conociendo HASH_SIZE, no hace falta delimitador '\0'
			word[i] = str[i];
			i++;
		}
		// El final del diccionari sera: word + (entradas * HASH_SIZE * sizeof(char)) no inclos.
		// Salto de palabra:
		word = word + (HASH_SIZE * sizeof(char));
	}
	cout << "Reading completed" << endl;
	fitxer.close();
	cout << "Dictionary closed" << endl;
	
	// Stats:
	cout << endl << "Numero de entradas existentes: " << entradas << endl;	
	//
	// End file reading and stored in memory.
	
	//
	//
	// Declaracions comuns:
	cout << endl << "Reserva de memoria para los resultados: distancias calculadas e índice para cada thread" << endl;
	unsigned int *out_idx;
	unsigned short *out_dist;
	CUDA_CALL( hipMallocManaged(&out_idx, out_gpu_len * sizeof(unsigned int)) );
	CUDA_CALL( hipMallocManaged(&out_dist, out_gpu_len * sizeof(unsigned short)) );
	
	//In file
	ifstream in_file;
	cout << endl << "Documento para tomar inputs: ";
	cin >> str;
	in_file.open(str.c_str());
	if(!in_file){
        cout << endl << str << " no obert correctament. EXIT." << endl;
        return EXIT_FAILURE; 
    }
	// Out file
	ofstream out_file;
	if(mode == correct_mode)
		out_file.open("corrected.txt", ofstream::out | ofstream::app);
	else
		out_file.open("report.txt", ofstream::out | ofstream::app);
	
    if(!out_file)
    {
        cout << endl << "Sortida no oberta/creada correctament. EXIT." << endl;
        return EXIT_FAILURE; 
    }
	cout << "Resultado en fichero: corrected.txt" << endl;
	
	// Variables auxiliars per correccio:
	char *query_word;
	CUDA_CALL( hipMallocManaged(&query_word, HASH_SIZE * sizeof(char)) );
	unsigned short _min;
	unsigned int launches = 0, corrected_w = 0, wrong_words = 0;
	char insp;
	//
	//
	// Consultes sobre memoria o correccio amb sa memoria
	//
	if(mode == correct_mode)
	{
		while ( !in_file.eof() && in_file.good() && out_file.good() )
		{
			i = 0;
			while(i < HASH_SIZE)
			{ // Get raw word
				insp = (char) in_file.peek();
				if( isalpha(insp) ) {
					in_file.get(query_word[i]);
				}
				else {
					query_word[i] = PADDING_CHAR;
				}
				i++;
			}
			// Consulta a memoria.
			k_levenshtein<<<num_blocks, threads_per_block>>>(query_word, first_word, entradas, out_idx, out_dist);
			CUDA_CALL( hipDeviceSynchronize() );
			launches++;

			// Encontramos el resultado de menor valor
			_min = 0;
			for(unsigned short k = 0; (k < out_gpu_len) && (out_dist[_min] != 0); k++) {
				if (out_dist[_min] > out_dist[k]) {
					_min = k;
				}
			}
			if(out_dist[_min] != 0)
				corrected_w++; 
			
			// Con el mínimo tenemos el puntero a la palabra que se escribe en out_file
			word = first_word + (out_idx[_min] * HASH_SIZE * sizeof(char));
			for(unsigned char k = 0; (k < HASH_SIZE) && (word[k] != PADDING_CHAR); k++) { out_file << word[k]; }
			
			// Escriure caracters entre paraules:
			while( !isalpha((char)in_file.peek()) && !in_file.eof() ) 
			{
				in_file.get(insp);
				out_file << insp;
				if(insp == '*') { wrong_words++; }
			}
			if((launches % 10000) == 0)
				cout << "Launches: " << launches << endl;
		}
	}
	else
	{
		while( !in_file.eof() && in_file.good() )
		{
			// Input de paraula per calcular distancia:
			str.clear();
			in_file >> str;
			out_file << "Q: " << str << " ";
			
			// Relleno de lo que nos entra...
			for(unsigned short letra = 0; letra < HASH_SIZE; letra++){
				if(letra < str.length()){
					query_word[letra] = str[letra];
				} else {
					query_word[letra] = (char)PADDING_CHAR;
				}
			}

			k_levenshtein<<<num_blocks, threads_per_block>>>(query_word, first_word, entradas, out_idx, out_dist);
			CUDA_CALL( hipDeviceSynchronize() );
			launches++;
			
			// Bucle de resultats...
			cout << endl << "Look for minimum dist result..." << endl;
			_min = 0;
			for(i = 0; i < out_gpu_len; i++){
				if(out_dist[i] <= out_dist[_min]){
					_min = i;
				}
			}

			// Apuntamos a la palabra seleccionada y al fichero
			word = first_word + ( out_idx[_min] * HASH_SIZE * sizeof(char) ); // Ponemos el puntero en la palabra que necesitamos...
			for(unsigned char k = 0; (k < HASH_SIZE) && (word[k] != PADDING_CHAR); k++) { out_file << word[k]; }
			out_file << " (" << out_dist[_min] << ")" << endl;
		}
	}
	
	cout << "Processing ended" << endl;
	// Tancam I/O files:
	in_file.close(); out_file.close();
	cout << "Archivos cerrados" << endl;
	
	//
	//
	//
	//
		
	CUDA_CALL( hipFree(first_word) );
	CUDA_CALL( hipFree(query_word) ); 
	CUDA_CALL( hipFree(out_idx)    );
	CUDA_CALL( hipFree(out_dist)   );

	cout << "cudaDeviceReset()..." << endl;
	CUDA_CALL( hipDeviceReset() );
	cout << "Device cleared" << endl;

	if(mode == correct_mode)
	{
		cout << endl << "======================== RUN INFO ========================" << endl;
		cout << " - Threads tot: " << out_gpu_len << endl;
		cout << " - Blocks:" << num_blocks << '\t' << "Threads: " << threads_per_block << endl << endl;
		cout << " - Words searched: " << launches << endl;
		cout << " - Total corrections: " << corrected_w << endl;
		cout << " - Real alterations: " << wrong_words << endl;
	}
	else
	{
		cout << endl << "======================== RUN INFO ========================" << endl;
		cout << " - Threads tot: " << out_gpu_len << endl;
		cout << " - Blocks:" << num_blocks << '\t' << "Threads: " << threads_per_block << endl << endl;
		cout << " - Words consulted: " << launches << endl;
	}
	
    return 0;
}


// Definimos correct_usage:
void correct_usage()
{
	cout << endl << "Author: Jover Mulet, Mateu. Contact @ mateu.jover@gmail.com" << endl;
	cout << "Electrical and Electronics Engineer from the U. of the Balearic Islands" << endl;
	cout << endl << "Setting the computational grid. Correct use for Ubuntu terminal command line options:" << endl;
	cout << " --grid [Threads per block (default: 256)] [Number of blocks(default: 4)] [--correct or --test(default)]" << endl;
	cout << "Check your GPGPU's specs for a better usage. Desired multiples of 32 for Threads per block." << endl;
		
	cout << "Displaying help, as it has been done. Just the parameter --help " << endl;
}